// SPDX-PackageName: "ACTS"
// SPDX-FileCopyrightText: 2016 CERN
// SPDX-License-Identifier: MPL-2.0

#pragma once

#include "Acts/Plugins/Cuda/Utilities/CpuScalar.hpp"

#include <iostream>
#include <memory>

#include "CudaUtils.cu"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

namespace Acts {

template <typename var_t>
class CpuScalar;

template <typename var_t>
class CudaScalar {
 public:
  CudaScalar() {
    ACTS_CUDA_ERROR_CHECK(hipMalloc((var_t**)&m_devPtr, sizeof(var_t)));
  }

  CudaScalar(var_t* scalar) {
    ACTS_CUDA_ERROR_CHECK(hipMalloc((var_t**)&m_devPtr, sizeof(var_t)));
    ACTS_CUDA_ERROR_CHECK(
        hipMemcpy(m_devPtr, scalar, sizeof(var_t), hipMemcpyHostToDevice));
  }

  CudaScalar(const var_t* scalar) {
    ACTS_CUDA_ERROR_CHECK(hipMalloc((var_t**)&m_devPtr, sizeof(var_t)));
    ACTS_CUDA_ERROR_CHECK(
        hipMemcpy(m_devPtr, scalar, sizeof(var_t), hipMemcpyHostToDevice));
  }

  ~CudaScalar() { ACTS_CUDA_ERROR_CHECK(hipFree(m_devPtr)); }

  var_t* get() { return m_devPtr; }

  void zeros() { hipMemset(m_devPtr, 0, sizeof(var_t)); }

 private:
  var_t* m_devPtr;
};
}  // namespace Acts

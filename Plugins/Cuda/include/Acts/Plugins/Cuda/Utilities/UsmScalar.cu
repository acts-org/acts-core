// SPDX-PackageName: "ACTS"
// SPDX-FileCopyrightText: 2016 CERN
// SPDX-License-Identifier: MPL-2.0

#pragma once

#include <iostream>
#include <memory>

#include "CudaUtils.cu"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

namespace Acts {

template <typename var_t>
class UsmScalar {
 public:
  UsmScalar() {
    ACTS_CUDA_ERROR_CHECK(hipMallocManaged((var_t**)&m_devPtr, sizeof(var_t)));
    hipDeviceSynchronize();
  }

  UsmScalar(var_t scalar) {
    ACTS_CUDA_ERROR_CHECK(hipMallocManaged((var_t**)&m_devPtr, sizeof(var_t)));
    hipDeviceSynchronize();
    m_devPtr[0] = scalar;
  }

  UsmScalar(var_t* scalar) {
    ACTS_CUDA_ERROR_CHECK(hipMallocManaged((var_t**)&m_devPtr, sizeof(var_t)));
    hipDeviceSynchronize();
    m_devPtr[0] = *scalar;
  }

  UsmScalar(const var_t* scalar) {
    ACTS_CUDA_ERROR_CHECK(hipMallocManaged((var_t**)&m_devPtr, sizeof(var_t)));
    hipDeviceSynchronize();
    m_devPtr[0] = *scalar;
  }

  ~UsmScalar() {
    hipDeviceSynchronize();
    ACTS_CUDA_ERROR_CHECK(hipFree(m_devPtr));
  }

  var_t* get() { return m_devPtr; }
  void set(var_t scalar) { m_devPtr[0] = scalar; }

 private:
  var_t* m_devPtr;
};
}  // namespace Acts

// SPDX-PackageName: "ACTS"
// SPDX-FileCopyrightText: 2016 CERN
// SPDX-License-Identifier: MPL-2.0

#pragma once

#include <iostream>

#include <hip/hip_runtime.h>


#define ACTS_CUDA_ERROR_CHECK(ans) \
  { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDAassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
